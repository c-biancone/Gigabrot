﻿
#include "hip/hip_runtime.h"


#include <cstdio>
#include <stdio.h>

void CPUFunction()
{
	printf("hello from the Cpu.\n");
}

__global__
void GPUFunction()
{
	printf("hello from the Gpu.\n");
}

int main()
{
	CPUFunction();

	GPUFunction<<<1, 1 >>>();

	hipDeviceSynchronize();

	return EXIT_SUCCESS;
}