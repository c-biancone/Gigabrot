#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <cstdio>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <complex>
#include <hip/hip_complex.h>
#include <math.h>

#define M_PI  3.14159265358979323846
#define M_LN2 0.69314718055994530942

// CUDA error check
#define cuda_check(call) \
	{\
	hipError_t res = (call);\
	if(res != hipSuccess) {\
		const char* err_str = hipGetErrorString(res);\
		fprintf(stderr, "%s (%d): %s in %s", __FILE__, __LINE__, err_str, #call);	\
		exit(-1);\
		}\
	}

/************************************* image variables ********************************************/
int pX, pY;
const int pXmax = 500; // 2 billion+ px each side should be enough resolution right???????
const int pYmax = 500; // for main antenna
const int iterationMax = 1000;
#define BS 256
/**************************************************************************************************/

/****************************** coordinate plane to be rendered ***********************************/
const double CxMin = -2.2;
const double CxMax = 0.8;
const double CyMin = -1.5;
const double CyMax = 1.5;
/**************************************************************************************************/

/**************************************** file stuff **********************************************/
double pixelWidth; // =(CxMax-CxMin)/pXmax;
double pixelHeight; // =(CyMax-CyMin)/pYmax;
const int maxColorComponentValue = 255; // rgb - SDR colorspace (8 bits per color)
FILE* fp;
char* filename = "mandelbrot.ppm";
// char * comment = "# "; // comment should start with #
/**************************************************************************************************/

/************************************* render parameters ******************************************/
unsigned char stripeDensity = 7; // higher is more dense
int i_skip = 1; // exclude (i_skip+1) elements from average
const double escapeRadius = 1000000; // big! (bail-out value)
double lnER;
/**************************************************************************************************/

double gpuTime = 0;

// number of threads PER ROW
int threadNum = pXmax;

/************************************ define complex type *****************************************/
struct _Complex {
	__host__ __device__ _Complex(float re, float im = 0) {
		this->re = re;
		this->im = im;
	}
	/** real and imaginary part */
	double re, im;
}; // struct complex
/********************************* define complex operators ***************************************/
inline __host__ __device__ _Complex operator+
(const _Complex& a, const _Complex& b) {
	return _Complex(a.re + b.re, a.im + b.im);
}
inline __host__ __device__ _Complex operator-
(const _Complex& a) {
	return _Complex(-a.re, -a.im);
}
inline __host__ __device__ _Complex operator-
(const _Complex& a, const _Complex& b) {
	return _Complex(a.re - b.re, a.im - b.im);
}
inline __host__ __device__ _Complex operator*
(const _Complex& a, const _Complex& b) {
	return _Complex(a.re * b.re - a.im * b.im, a.im * b.re + a.re * b.im);
}
inline __host__ __device__ float abs2(const _Complex& a) {
	return a.re * a.re + a.im * a.im;
}
inline __host__ __device__ _Complex operator/
(const _Complex& a, const _Complex& b) {
	float invabs2 = 1 / abs2(b);
	return _Complex((a.re * b.re + a.im * b.im) * invabs2,
		(a.im * b.re - b.im * a.re) * invabs2);
}  // operator/
/**************************************************************************************************/


/**
 * Function: get_c
 * ---------------
 * Gets the corresponding coordinate point for the location of a pixel.
 *
 * Inputs:
 *  iX: number of x-iteration
 *  iY: number of y-iteration
 *
 * Returns:
 *  Complex double form of the coordinate.
 */

_Complex get_c(int iX, int iY) {
	double Cx, Cy;
	Cy = CyMax - iY * pixelHeight;
	//if (fabs(Cy)< pixelHeight/3.0) Cy=0.0; // main antenna
	Cx = CxMin + iX * pixelWidth;
	return (Cx, Cy);
}

/**
 * Function: c_dot
 * ---------------
 * Computes the dot product of 2 complex double vectors.
 *
 * Inputs:
 *  a: vector 1
 *  b: vector 2
 *
 * Returns:
 *  The dot product in double form.
 */
double c_dot(_Complex a, _Complex b) {
	return a.re * b.re + a.im * b.im;
}

/**
 * Function: c_arg
 * ---------------
 * Computes the argument of a given complex vector.
 *
 * Inputs:
 *  a: vector 1
 *
 * Returns:
 *  The argument in double form.
 */
double c_arg(_Complex a) {
	return atan2(a.im, a.re);
}
/**
 * Function: c_abs
 * ---------------
 * Computes the magnitude of a given vector.
 *
 * Inputs:
 *  a: vector 1
 *
 * Returns:
 *  The magnitude in double form.
 */
double c_abs(_Complex a) {
    return sqrt((a.re * a.re) + (a.im * a.im));
}
/**
 * Function: get_t
 * ---------------
 * Addend function used to get the value of A for stripe-average coloring method.
 * https://en.wikibooks.org/wiki/Fractals/Iterations_in_the_complex_plane/stripeAC
 *
 * Inputs:
 *  z: complex number
 *
 * Returns:
 *  Double number
 */
double getT(_Complex z) {
	return 0.5 + 0.5 * sin(stripeDensity * c_arg(z));
}
/**
 * Function: setup
 * ---------------
 * Sets up the .ppm file stream and parameters.
 *
 * Inputs:
 *  NULL
 *
 * Returns:
 *  NULL
 */
void setup() {
	pixelWidth = (CxMax - CxMin) / pXmax;
	pixelHeight = (CyMax - CyMin) / pYmax;
	lnER = log(escapeRadius);

	// create new ppm6 file, give it a name, and open it in binary mode
	fp = fopen(filename, "wb");
	// write ASCII header to the file
	fprintf(fp, "P6\n %d\n %d\n %d\n", pXmax, pYmax, maxColorComponentValue);
}
/**
 * Function: info
 * --------------
 * Provides debugging information from the file.
 *
 * Inputs:
 *  NULL
 *
 * Returns:
 *  NULL
 */
void info() {
	double distortion;
	// width/height
	double pixelsAspectRatio = (double)pXmax / pYmax;
	double worldAspectRatio = (CxMax - CxMin) / (CyMax - CyMin);
	// printf("pixelsAspectRatio = %.16f \n", pixelsAspectRatio);
	// printf("worldAspectRatio = %.16f \n", worldAspectRatio);
	distortion = pixelsAspectRatio - worldAspectRatio;
	printf("distortion = %.16f (should be zero!)\n", distortion);
	// printf("bailout value = Escape Radius = %.0f \n", escapeRadius);
	// printf("iterationMax = %d \n", iterationMax);
	// printf("i_skip = %d = number of skipped elements ( including t0 )= %d \n", i_skip, i_skip+1);
	printf("file %s saved.\n", filename);
}
/**
 * Function: close
 * ---------------
 * Closes file stream and calls debugging info function.
 *
 * Inputs:
 *  NULL
 *
 * Returns:
 *  NULL
 */
void close() {
	fclose(fp);
	info();
}

__device__ int colorize(_Complex c, unsigned char* row, int iX, int iMax) {
    /** global **/
    unsigned char b; // color
    int i; // iteration
    /** normal map **/
     _Complex Z = 0.0; // initial value for iteration Z0
     _Complex dC = 0.0; // derivative with respect to c
    double reflection = FP_ZERO; // inside
    double h2 = 1.5; // height factor of the incoming light
    double angle = 45.0 / 360.0; // incoming direction of light in turns (change 1st #)
    _Complex v = 0; //cexp(2.0 * angle * M_PI * I); // unit 2D vector in this direction
    v.re = cos(2.0 * angle * M_PI); // this works because the exponent is purely imaginary
    v.im = sin(2.0 * angle * M_PI);
    _Complex u = 0.0; // normal
    /** arg vars **/
    double A = 0.0; // A(n)
    double prevA = 0.0; // A(n-1)
    double R;
    double d; // smooth iteration count
    double de; // boundary descriptor


    /** do the compute **/
    for (i = 0; i < iMax; i++) {
        dC = 2.0 * dC * Z + 1.0;
        Z = Z * Z + c;
        if (i > i_skip) A += getT(Z);
        R = c_abs(Z);
        /* shape checking algorithm
           skips iterating points within the main cardioid and secondary bulb, otherwise these
           would all hit the max iterations
           removes about 91% of the set from iteration
           REMOVE THIS IF NOT RENDERING THE ENTIRE SET - more computation per iteration if the main
           cardioid and secondary bulb are not shown onscreen */
        double q = ((c.re - 0.25) * (c.im - 0.25)) + (c.im * c.im);
        double cardioid = 0.25 * c.im * c.im;
        double bulb = 0.0625;
        if ((c.re * c.re + 2 * c.re + 1 + c.im * c.im) < bulb || (q * (q +
            (c.re - 0.25)) < cardioid)) {
            break;
        }
        /** get normal map **/
        if (R > escapeRadius) { // exterior of M set
            u = Z / dC;
            u = u / c_abs(u);
            reflection = c_dot(u, v) + h2;
            reflection = reflection / (1.0 + h2); // rescale so that t does not get bigger than 1
            if (reflection < 0.0) reflection = 0.0;
            break;
        }
        prevA = A; // save value for interpolation
    }
    /** get striping **/
    if (i == iMax)
        A = -1.0; // interior
    else { // exterior
        de = 2 * R * log(R) / c_abs(dC);
        int thin = 3; // thinness of the border
        if (de < (pixelWidth / thin)) A = FP_ZERO; //  boundary
        else {
            // computing interpolated average
            A /= (i - i_skip); // A(n)
            prevA /= (i - i_skip - 1); // A(n-1)
            // smooth iteration count
            d = i + 1 + log(lnER / log(R)) / M_LN2;
            d = d - (int)d; // only fractional part = interpolation coefficient
            // linear interpolation
            A = d * A + (1.0 - d) * prevA;
        }
    }

    /** assign pixel color values **/
    int subPixel = 3 * iX;
    if (reflection == FP_ZERO) { // interior of Mandelbrot set = black
        /* ppm files have pixels situated as groups of 3 ASCII chars in a row; the columns of the
           image file will be 3x as numerous as the rows
           attempting to store the image rows as a vector in memory and write to the file 1 row
           at a time */
        row[subPixel] = 0;
        row[subPixel + 1] = 0;
        row[subPixel + 2] = 0;
    }

    // exterior of Mandelbrot set -> normal
    else { // multiply the underlying stripe gradient by the reflectivity map
        if (A == FP_ZERO) b = 255; // boundary
        else b = (unsigned char)((254 - (100 * A)) * reflection); // set color bounds for striping

        row[subPixel] = b;
        row[subPixel + 1] = b;
        row[subPixel + 2] = b;
    }
    return 0;
}

int main() {

    _Complex c = 0.0;
    unsigned char row[pXmax * 3];

    setup();

    printf("Rendering row by row");

    for (pY = 0; pY < pYmax; pY++) {
        for (pX = 0; pX < pXmax; pX++) {

        }
            
        
    }



    printf("Render took %.9 lf seconds.\n", gpuTime);
}



void CPUFunction()
{
	printf("hello from the Cpu.\n");
}

__global__
void GPUFunction()
{
	printf("hello from the Gpu.\n");
}

int main()
{
	CPUFunction();

	GPUFunction<<<1, 1 >>>();

	hipDeviceSynchronize();

	return EXIT_SUCCESS;
}